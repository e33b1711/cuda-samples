#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>

#include "aux.h"

//TODO line interp

__device__ uchar4 mapping(unsigned short hist_count, const int n_spec ){

    assert(hist_count<=n_spec);

    //0 => 0 0 0 0
    if (hist_count==0) return uchar4(0,0,0,0);

    // 1 => 0.0 / 0
    // n_spec => 1.0 / 510
    float color_index = log(hist_count) / log(n_spec);
    int c_index = int(510.0 * color_index);

    //index: 0-255
    //       0-255
    //       255-0
    //       0
    if(c_index<256){
        return uchar(c_index, 255-c_index, 0, 0);
    }
    //index: 256-510
    //       254-0
    //       0
    //       1-255
    if(c_index<510){
        return uchar(510-c_index, 0, c_index-255, 0);
    }

    return uchar(255, 255, 255, 0);
}


__global__ void polchrome_kernel(const float2* f_domain, uchar4 *ptr, const short n_bins, const int n_spec) {
    //one block per bin
    //32 threads per block
    //we are heavly memory constricted

    assert(gridDim.x == 1024);
    assert(blockDim.x == 32);

    const int bin_idx = blockIdx.x;
    const int thread_idx = threadIdx.x;
    const int num_threads = 32;
    const unsigned int height = 512;

    __shared__ unsigned short hist_column[num_threads*height];

    for(int y_ind=0; y_ind<height; y_ind++) hist_column[thread_idx* height + y_ind] = 0;

    int idx= bin_idx + thread_idx*n_bins;
    while (idx < n_bins*n_spec) {

        float2 fd = f_domain[idx];
        float abs_db = 20.0f * log10(sqrtf(fd.x * fd.x + fd.y * fd.y));
        const float scale = 2.0f;
        int unsigned y_ind = int( scale * abs_db + height/2) % height;

        hist_column[thread_idx*height + y_ind] ++;

        idx += num_threads*n_bins;
    }

    __syncthreads();

    int i = num_threads/2;
    while (i != 0) {
        if (thread_idx < i){
            for(int y_ind=0; y_ind<height; y_ind++) hist_column[thread_idx* height + y_ind] += hist_column[(thread_idx + i)*height+y_ind];
        }
        __syncthreads();
        i /= 2;
    }

    if (thread_idx == 0){
       for(int h=0; h<height; h++) hist[bin_idx  + h * n_bins] = mapping(hist_column[h], n_spec);
    }

    assert(mapping(0)==uchar4(0,0,0,0))
    assert(mapping(1)==uchar4(0,1,0,0))
    assert(mapping(n_spec)==uchar4(0,0,1,0))
}


void polchrome(float2* f_domain, unsigned short* hist, const int block_len, const int n_blocks){
    // Timing start
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    const int blockSize = 32;
    const int numBlocks = n_bins;
    polchrome_kernel<<<numBlocks, blockSize>>>(f_domain, hist, block_len, n_blocks);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // Timing end
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    static int disp_count = 0;
    if((disp_count++)%100 == 0) printf("FFT postproc time: %.3f ms\n", ms);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
}