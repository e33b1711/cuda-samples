#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>

#include "aux.h"
#include "fft.h"

__device__ uchar4 mapping(unsigned short hist_count, const int n_spec ){

    assert(hist_count<=n_spec);

    //0 => 0 0 0 0
    if (hist_count==0) return uchar4(0,0,0,0);

    // 1 => 0.0 / 0
    // n_spec => 1.0 / 510
    float color_index = log(hist_count) / log(n_spec);
    int c_index = int(510.0 * color_index);

    //index: 0-255
    //       0-255
    //       255-0
    //       0
    if(c_index<256){
        return uchar(c_index, 255-c_index, 0, 0);
    }
    //index: 256-510
    //       254-0
    //       0
    //       1-255
    if(c_index<510){
        return uchar(510-c_index, 0, c_index-255, 0);
    }

    return uchar(255, 255, 255, 0);
}

__device__ float db_abs(float2 fd) {
    return 20.0f * log10(sqrtf(fd.x * fd.x + fd.y * fd.y));
}


__device__ void line_interp(int* y_max, int* y_max, const float2* d_signal, const int x, const int height, const float scale){
    float abs_x_mid = db_abs(d_signal[x]);
    float abs_x_left = (x>0)? db_abs(d_signal[x-1]) : abs_x_mid;
    float abs_x_right = (x<width-1)? db_abs(d_signal[x+1]) : abs_x_mid;
    int y_mid = int( scale * abs_x_mid + height/2);
    int left_y = int( 0.5 * scale * (abs_x_left + abs_x_mid) + height/2);
    int right_y = int( 0.5 * scale * (abs_x_right + abs_x_mid) + height/2);
    y_max = max(max(left_y, y_mid),right_y);
    y_min = min(min(left_y, y_mid),right_y);
}


__global__ void polchrome_kernel(const float2* f_domain, uchar4 *ptr, const short n_bins, const int n_spec) {
    //one block per bin
    //32 threads per block
    //we are heavly memory constricted

    assert(gridDim.x == 1024);
    assert(blockDim.x == 32);

    const int bin_idx = blockIdx.x;
    const int thread_idx = threadIdx.x;
    const int num_threads = 32;
    const unsigned int height = 512;

    __shared__ short hist_column[num_threads*height];

    for(int y_ind=0; y_ind<height; y_ind++) hist_column[thread_idx* height + y_ind] = 0;

    int idx= bin_idx + thread_idx*n_bins;
    while (idx < n_bins*n_spec) {

        const float scale = 2.0f;
        int y_min, y_max;
        line_interp(y_min, y_max, d_signal, x, height, scale);

        hist_column[thread_idx*height + y_min] ++;
        hist_column[thread_idx*height + y_max] --

        idx += num_threads*n_bins;
    }

    __syncthreads();

    //collect cache fom threads / reduce
    int i = num_threads/2;
    while (i != 0) {
        if (thread_idx < i){
            for(int y_ind=0; y_ind<height; y_ind++) hist_column[thread_idx* height + y_ind] += hist_column[(thread_idx + i)*height+y_ind];
        }
        __syncthreads();
        i /= 2;
    }

    //integrate
    if (thread_idx == 0){
       for(int h=1; h<height; h++){
            hist[bin_idx  + h*n_bins] += hist[bin_idx  + (h-1)*n_bins];
    }

    //map
    if (thread_idx == 0){
       for(int h=0; h<height; h++) hist[bin_idx  + h * n_bins] = mapping(hist_column[h], n_spec);
    }

    //TODO delete me!! Make proper unit tests
    assert(mapping(0)==uchar4(0,0,0,0))
    assert(mapping(1)==uchar4(0,1,0,0))
    assert(mapping(n_spec)==uchar4(0,0,1,0))
}


void polchrome(float2* f_domain, unsigned short* hist, const int block_len, const int n_blocks){
    // Timing start
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    const int blockSize = 32;
    const int numBlocks = n_bins;
    polchrome_kernel<<<numBlocks, blockSize>>>(f_domain, hist, block_len, n_blocks);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // Timing end
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    static int disp_count = 0;
    if((disp_count++)%100 == 0) printf("FFT postproc time: %.3f ms\n", ms);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
}