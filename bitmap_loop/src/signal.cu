#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>
#include <hipfft/hipfft.h>

#include "aux.h"


__global__ void generatePhasorSignal(float2* signal, int length, float omega, float phi, float noiseVariance, unsigned long long seed, int spike_index) {
    hiprandState state;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state);
    while (idx < length) {
        float noiseReal = hiprand_normal(&state) * noiseVariance;
        float noiseImag = hiprand_normal(&state) * noiseVariance;
        float angle = omega * idx + phi;
        signal[idx].x = cosf(angle) + noiseReal; // Real part
        signal[idx].y = sinf(angle) + noiseImag; // Imaginary part
        if (idx==(spike_index % length)){
            signal[idx].x = 20.0;
            signal[idx].y = 20.0;
        }
        idx += blockDim.x  * gridDim.x;
    }
}


void generate_signal(hipStream_t stream, float2* d_signal, const float phi, const int length, const int frame){
    // Timing start
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int blockSize = 256;
    float omega = 0.1f * 3.14159265359f; // 5 cycles over the signal
    int numBlocks = 256;
    float noiseVariance = 0.5f;
    generatePhasorSignal<<<numBlocks, blockSize, 0, stream>>>(d_signal, length, omega, phi, noiseVariance, (unsigned long long) frame, rand());


    // Timing end
    hipEventRecord(stop, 0);
    //hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    static int disp_count = 0;
    if((disp_count++)%100 == 0) printf("Signal generation time: %.3f ms\n", ms);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
