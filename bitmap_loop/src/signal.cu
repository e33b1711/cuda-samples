#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>
#include <hipfft/hipfft.h> // Add this include

#include "aux.h"

__global__ void addAWGN(float2* signal, float noiseVariance, int length, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        float noiseReal = hiprand_normal(&state) * sqrt(noiseVariance);
        float noiseImag = hiprand_normal(&state) * sqrt(noiseVariance);
        signal[idx].x += noiseReal;
        signal[idx].y += noiseImag;
    }
}

__global__ void generatePhasorSignal(float2* signal, int length, float omega, float phi) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        float angle = omega * idx + phi;
        signal[idx].x = cosf(angle); // Real part
        signal[idx].y = sinf(angle); // Imaginary part
    }
}

void generate_signal(float2* d_signal, const float phi, const int length, const int frame){
    int blockSize = 256;
    float omega = 0.1f * 3.14159265359f; // 5 cycles over the signal
    int numBlocks = (length + blockSize - 1) / blockSize;
    generatePhasorSignal<<<numBlocks, blockSize>>>(d_signal, length, omega, phi);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    float noiseVariance = 0.5f;
    addAWGN<<<numBlocks, blockSize>>>(d_signal, noiseVariance, length, (unsigned long long) frame);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}

void run_fft(float2* t_domain, float2* f_domain, int length, int count) {
    hipfftHandle plan;
    hipfftResult result;

    // Timing start
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Create a 1D FFT plan for complex-to-complex (single precision)
    result = hipfftPlan1d(&plan, length, HIPFFT_C2C, count);
    assert(result == HIPFFT_SUCCESS);

    // Execute FFT (forward transform)
    result = hipfftExecC2C(plan, (hipfftComplex*)t_domain, (hipfftComplex*)f_domain, HIPFFT_FORWARD);
    assert(result == HIPFFT_SUCCESS);

    // Timing end
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    printf("FFT calculation time: %.3f ms\n", ms);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipfftDestroy(plan);
}

__global__ void fft_detector(float2* f_domain, float* f_max, float* f_min, float* f_mean, int length, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= length) return;
    float max_v = -1e99f;
    float min_v = 1e99f;
    float mean_v = 0.0f;
    int idx_up = idx;
    while (idx_up < length*count) {
        float abs_v = sqrt( pow(f_domain[idx_up].x, 2) + pow(f_domain[idx_up].y, 2) ); 
        max_v = max(max_v, abs_v);
        min_v = min(min_v, abs_v);
        mean_v += abs_v; 
        idx_up += length;
    }
    f_max[idx] = max_v;
    f_min[idx] = min_v;
    f_mean[idx] = mean_v / count;
}

void fft_postproc(float2* f_domain, float* f_max, float* f_min, float* f_mean, const int length, const int count){
    int blockSize = 256;
    int numBlocks = (length + blockSize - 1) / blockSize;
    fft_detector<<<numBlocks, blockSize>>>(f_domain, f_max, f_min, f_mean, length, count);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}