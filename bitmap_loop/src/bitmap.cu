#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>
#include <hipfft/hipfft.h>

#include "aux.h"

__device__ float db_abs(float d_signal) {
    return 20.0f * log10(d_signal);
}

__global__ void fill_bitmap_spec(uchar4 *ptr, int width, int height, float* d_signal, int color, bool clear) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;

    if (clear){
        ptr[idx].x = 0;
        ptr[idx].y = 0;
        ptr[idx].z = 0;
        ptr[idx].w = 0;
    }


    const float scale = 2.0f;

    float abs_x_mid = db_abs(d_signal[x]);
    float abs_x_left = (x>0)? db_abs(d_signal[x-1]) : abs_x_mid;
    float abs_x_right = (x<width-1)? db_abs(d_signal[x+1]) : abs_x_mid;

    int y_mid = int( scale * abs_x_mid + height/2);
    int left_y = int( 0.5 * scale * (abs_x_left + abs_x_mid) + height/2);
    int right_y = int( 0.5 * scale * (abs_x_right + abs_x_mid) + height/2);
    int y_max = max(max(left_y, y_mid),right_y);
    int y_min = min(min(left_y, y_mid),right_y);

    if (y <= y_max and y >= y_min){
        if (color==0) ptr[idx].x = 255;
        if (color==1) ptr[idx].z = 255;
        if (color==2) ptr[idx].y = 255;
    }
}