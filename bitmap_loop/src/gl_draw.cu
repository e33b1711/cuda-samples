#include "hip/hip_runtime.h"
#include "gl_draw.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdio.h>

#include "aux.h"


hipGraphicsResource *cuda_pbo_resource;
GLuint pbo = 0, tex = 0;


void initGLUT(int *argc, char **argv, void (*cleanupFunc)(), int height, int width) {
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(width, height);
    glutCreateWindow("CUDA Bitmap via GLUT");
    GLenum err = glewInit();
    if (err != GLEW_OK) {
        fprintf(stderr, "GLEW init failed: %s\n", glewGetErrorString(err));
        exit(1);
    }
    atexit(cleanupFunc);
}


void initPixelBuffer(GLuint *pbo, GLuint *tex, hipGraphicsResource **cuda_pbo_resource, int height, int width) {
    glGenBuffers(1, pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, *pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * 4, 0, GL_DYNAMIC_DRAW);

    hipGraphicsGLRegisterBuffer(cuda_pbo_resource, *pbo, cudaGraphicsMapFlagsWriteDiscard);

    glGenTextures(1, tex);
    glBindTexture(GL_TEXTURE_2D, *tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glEnable(GL_TEXTURE_2D);
}


void drawGL(GLuint pbo, GLuint tex, int width, int height) {
    glClear(GL_COLOR_BUFFER_BIT);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

    glBegin(GL_QUADS);
    glTexCoord2f(0, 0); glVertex2f(-1, -1);
    glTexCoord2f(1, 0); glVertex2f(1, -1);
    glTexCoord2f(1, 1); glVertex2f(1, 1);
    glTexCoord2f(0, 1); glVertex2f(-1, 1);
    glEnd();

    glutSwapBuffers();
}


void dummy_display() {}


void draw_cleanup() {
    hipGraphicsUnregisterResource(cuda_pbo_resource);
    glDeleteBuffers(1, &pbo);
    glDeleteTextures(1, &tex);
}


void draw_init(const int height, const int width, int argc, char **argv){
    initGLUT(&argc, argv, draw_cleanup, height, width);
    glutDisplayFunc(dummy_display); // Register dummy display callback
    initPixelBuffer(&pbo, &tex, &cuda_pbo_resource, height, width);

}


void draw_loop(uchar4* bitmap, const int width, const int height){

    uchar4 *dptr;
    size_t num_bytes;
    CUDA_SAFE_CALL(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
    CUDA_SAFE_CALL(hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes, cuda_pbo_resource));
    CUDA_SAFE_CALL(hipMemcpy(dptr, bitmap, width * height * sizeof(uchar4), hipMemcpyDeviceToDevice););
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipGraphicsUnmapResources(1, &cuda_pbo_resource));
    drawGL(pbo, tex, width, height);
    
    glutMainLoopEvent();
}