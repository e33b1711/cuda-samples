#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>
#include <hipfft/hipfft.h>

#include "aux.h"
#include "bitmap.h"


void run_fft(float2* t_domain, float2* f_domain, int length, int count) {
    static hipfftHandle plan;
    static bool init = true;
    hipfftResult result;

    // Create a 1D FFT plan for complex-to-complex (single precision)
    if (init){
        result = hipfftPlan1d(&plan, length, HIPFFT_C2C, count);
        assert(result == HIPFFT_SUCCESS);
    }

     // Timing start
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Execute FFT (forward transform)
    result = hipfftExecC2C(plan, (hipfftComplex*)t_domain, (hipfftComplex*)f_domain, HIPFFT_FORWARD);
    assert(result == HIPFFT_SUCCESS);

    // Timing end
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    static int disp_count = 0;
    if((disp_count++)%100 == 0) printf("FFT calculation time: %.3f ms\n", ms);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipfftDestroy(plan);
}


__global__ void fft_detector(const float2* f_domain, float* f_max, float* f_min, float* f_mean, const int block_len, const int n_blocks) {

    const int bin_idx = blockIdx.x;
    const int thread_idx = threadIdx.x;
    const int num_threads = blockDim.x;
    const int num_threads_max = 1024;
    assert(num_threads <= num_threads_max);

    __shared__ float max_cache[num_threads_max];
    __shared__ float min_cache[num_threads_max];
    __shared__ float mean_cache[num_threads_max];

    float max_v = -1e99f;
    float min_v = 1e99f;
    float mean_v = 0.0f;
    int idx= bin_idx + thread_idx*block_len;
    while (idx < block_len*n_blocks) {
        float2 fd = f_domain[idx];
        float abs_v = sqrtf(fd.x * fd.x + fd.y * fd.y);
        max_v = max(max_v, abs_v);
        min_v = min(min_v, abs_v);
        mean_v += abs_v;

        idx += block_len*num_threads;
    }
    max_cache[thread_idx] = max_v;
    min_cache[thread_idx] = min_v;
    mean_cache[thread_idx] = mean_v;

    __syncthreads();

    int i = num_threads/2;
    while (i != 0) {
        if (thread_idx < i){
            max_cache[thread_idx] = max(max_cache[thread_idx], max_cache[thread_idx + i]);
            min_cache[thread_idx] = min(min_cache[thread_idx], min_cache[thread_idx + i]);
            mean_cache[thread_idx] += mean_cache[thread_idx + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (thread_idx == 0){
       f_max[bin_idx] = max_cache[0];
       f_min[bin_idx] = min_cache[0];
       f_mean[bin_idx] = mean_cache[0] / n_blocks;
    }
}


__device__ float db_abs(float d_signal) {
    return 20.0f * log10(d_signal);
}


__device__ void line_interp(int* y_max, int* y_max, const float* d_signal, const int x, const int height, const float scale){
    float abs_x_mid = db_abs(d_signal[x]);
    float abs_x_left = (x>0)? db_abs(d_signal[x-1]) : abs_x_mid;
    float abs_x_right = (x<width-1)? db_abs(d_signal[x+1]) : abs_x_mid;
    int y_mid = int( scale * abs_x_mid + height/2);
    int left_y = int( 0.5 * scale * (abs_x_left + abs_x_mid) + height/2);
    int right_y = int( 0.5 * scale * (abs_x_right + abs_x_mid) + height/2);
    y_max = max(max(left_y, y_mid),right_y);
    y_min = min(min(left_y, y_mid),right_y);
}


__global__ void fill_bitmap_spec(uchar4 *ptr, int width, int height, float* d_signal, int color, bool clear) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;

    if (clear){
        ptr[idx].x = 0;
        ptr[idx].y = 0;
        ptr[idx].z = 0;
        ptr[idx].w = 0;
    }

    const float scale = 2.0f;
    int y_min, y_max;
    line_interp(y_min, y_max, d_signal, x, height, scale);

    if (y <= y_max and y >= y_min){
        if (color==0) ptr[idx].x = 255;
        if (color==1) ptr[idx].z = 255;
        if (color==2) ptr[idx].y = 255;
    }
}


void fft_postproc(float2* f_domain, uchar4* bitmap, const int block_len, const int n_blocks, int width, int height){
    // Timing start
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    static float* f_max = nullptr;
    static float* f_min = nullptr;
    static float* f_mean = nullptr;
    static bool init = true;

    if(init){
        init = false;
        CUDA_SAFE_CALL(hipMalloc(&f_max, block_len * n_blocks * sizeof(float)));
        CUDA_SAFE_CALL(hipMalloc(&f_min, block_len * n_blocks * sizeof(float)));
        CUDA_SAFE_CALL(hipMalloc(&f_mean, block_len * n_blocks * sizeof(float)));
    }

    const int blockSize = 1024;
    const int numBlocks = n_bins;
    fft_detector<<<numBlocks, blockSize>>>(f_domain, f_max, f_min, f_mean, block_len, n_blocks);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    <<<numBlocks, blockSize>>>fill_bitmap_spec(bitmap, width, height, f_max, 1, false);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    <<<numBlocks, blockSize>>>fill_bitmap_spec(bitmap, width, height, f_min, 1, false);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    const int blockSize = 1024;
    const int numBlocks = n_bins;
    <<<numBlocks, blockSize>>>fill_bitmap_spec(bitmap, width, height, f_mean, 1, false);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // Timing end
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    static int disp_count = 0;
    if((disp_count++)%100 == 0) printf("FFT postproc time: %.3f ms\n", ms);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
}