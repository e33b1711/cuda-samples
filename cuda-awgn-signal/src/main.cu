#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "signal.h"
#include "draw.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

const int SIGNAL_LENGTH = 1024;

__global__ void generateComplexSignal(float2* signal, int length, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        signal[idx].x = hiprand_normal(&state); // Real part
        signal[idx].y = hiprand_normal(&state); // Imaginary part
    }
}

__global__ void addAWGN(float2* signal, float noiseVariance, int length, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        float noiseReal = hiprand_normal(&state) * sqrt(noiseVariance);
        float noiseImag = hiprand_normal(&state) * sqrt(noiseVariance);
        signal[idx].x += noiseReal;
        signal[idx].y += noiseImag;
    }
}

int main() {
    float2* d_signal = nullptr;
    CUDA_CHECK(hipMalloc(&d_signal, SIGNAL_LENGTH * sizeof(float2)));

    int blockSize = 256;
    int numBlocks = (SIGNAL_LENGTH + blockSize - 1) / blockSize;
    generateComplexSignal<<<numBlocks, blockSize>>>(d_signal, SIGNAL_LENGTH, 1234ULL);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    float noiseVariance = 0.1f;
    addAWGN<<<numBlocks, blockSize>>>(d_signal, noiseVariance, SIGNAL_LENGTH, 1234ULL);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Only this call is needed for drawing:
    displaySignalWithCudaInterop(d_signal, SIGNAL_LENGTH);

    CUDA_CHECK(hipFree(d_signal));
    return 0;
}