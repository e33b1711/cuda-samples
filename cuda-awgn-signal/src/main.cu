#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "draw.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

const int SIGNAL_LENGTH = 1024*1024;

__global__ void addAWGN(float2* signal, float noiseVariance, int length, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        float noiseReal = hiprand_normal(&state) * sqrt(noiseVariance);
        float noiseImag = hiprand_normal(&state) * sqrt(noiseVariance);
        signal[idx].x += noiseReal;
        signal[idx].y += noiseImag;
    }
}

__global__ void generatePhasorSignal(float2* signal, int length, float omega, float phi) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        float angle = omega * idx + phi;
        signal[idx].x = cosf(angle); // Real part
        signal[idx].y = sinf(angle); // Imaginary part
    }
}

int main() {
    float2* d_signal = nullptr;
    CUDA_CHECK(hipMalloc(&d_signal, SIGNAL_LENGTH * sizeof(float2)));

    int blockSize = 256;
    int numBlocks = (SIGNAL_LENGTH + blockSize - 1) / blockSize;

    float omega = 2.0f * 3.14159265359f * 5.0f / SIGNAL_LENGTH; // 5 cycles over the signal
    float phi = 0.0f;
    generatePhasorSignal<<<numBlocks, blockSize>>>(d_signal, SIGNAL_LENGTH, omega, phi);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    float noiseVariance = 0.01f;
    addAWGN<<<numBlocks, blockSize>>>(d_signal, noiseVariance, SIGNAL_LENGTH, 1234ULL);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    displaySignalWithCudaInterop(d_signal, SIGNAL_LENGTH);

    CUDA_CHECK(hipFree(d_signal));
    return 0;
}