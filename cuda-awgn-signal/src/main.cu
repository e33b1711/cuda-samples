
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "signal.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

const int SIGNAL_LENGTH = 1024;

__global__ void generateComplexSignal(float2* signal, int length, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        signal[idx].x = hiprand_normal(&state); // Real part
        signal[idx].y = hiprand_normal(&state); // Imaginary part
    }
}

__global__ void addAWGN(float2* signal, float noiseVariance, int length, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        float noiseReal = hiprand_normal(&state) * sqrt(noiseVariance);
        float noiseImag = hiprand_normal(&state) * sqrt(noiseVariance);
        signal[idx].x += noiseReal;
        signal[idx].y += noiseImag;
    }
}

int main() {
    float2* d_signal;
    size_t size = SIGNAL_LENGTH * sizeof(float2);
    CUDA_CHECK(hipMalloc(&d_signal, size));

    // Generate complex signal
    int blockSize = 256;
    int numBlocks = (SIGNAL_LENGTH + blockSize - 1) / blockSize;
    generateComplexSignal<<<numBlocks, blockSize>>>(d_signal, SIGNAL_LENGTH, 1234ULL);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Add AWGN
    float noiseVariance = 0.1f;
    addAWGN<<<numBlocks, blockSize>>>(d_signal, noiseVariance, SIGNAL_LENGTH, 1234ULL);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Here you would typically call a function to draw the signal
    // drawSignal(d_signal, SIGNAL_LENGTH);

    CUDA_CHECK(hipFree(d_signal));
    return 0;
}